// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#include <ff/bls12-381.hpp>

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_t affine_t;
typedef fr_t scalar_t;

#include <msm/pippenger.cuh>

#ifndef __CUDA_ARCH__

msm_t<bucket_t, point_t, affine_t, scalar_t> *pr_msm = nullptr;

extern "C"
void prepare_msm(const affine_t points[], size_t npoints) {
    pr_msm = new msm_t<bucket_t, point_t, affine_t, scalar_t>{points, npoints};
}

extern "C"
RustError mult_prepared_pippenger(point_t* out, size_t npoints, const scalar_t scalars[], bool mont) {
    return pr_msm->invoke(*out, slice_t<scalar_t>{scalars, npoints}, mont);
}

extern "C"
RustError mult_pippenger_mont(point_t* out, const affine_t points[], size_t npoints,
                                       const scalar_t scalars[])
{
    return mult_pippenger<bucket_t>(out, points, npoints, scalars, true);
}

extern "C"
RustError mult_pippenger(point_t* out, const affine_t points[], size_t npoints,
                                       const scalar_t scalars[])
{
    return mult_pippenger<bucket_t>(out, points, npoints, scalars, false);
}
#endif